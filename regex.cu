#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

extern "C" {
#include "regex.h"
#include "nfa.h"
}

#define MAX_FILE_SIZE 1 << 30
const int CHUNK = 1 << 14;
const int MAX_CONTEXT_SIZE = 500;
const int N_RESULTS = 50;
const int MAX_THREADS_PER_BLOCK = 1024;

typedef struct search_result {
  char* context;
  int line;
} res;

__global__ void regex_kernel(char** contents, res*** results, const re_t __restrict__ pattern, int file_no){
  int res_idx = 0;
  int line = 1;

  /* Local variables that keep track of the start and end of the context */
  /* TODO: out_before: needs to be initialized by going back until previous newline is found */
  int out_before = -1;
  uint8_t matched = 0;

  /* Read the ith file, check for pattern and write to result */
  char* start = &(contents[file_no][threadIdx.x * CHUNK]);
  res* result_loc = &(results[file_no][threadIdx.x][0]);
  char c;

  int i;
  for(i = 0; i < threadIdx.x * CHUNK && *(start-i) != '\n' && *(start-i) != '\0'; i++);
  out_before = -1 * i - (i == 0);

  for(i = 0; i < CHUNK && ((c = *(start + i)) != '\0'); i++){
	line += (c == '\n');

	if(matched && (c == '\n')){
	  /* Copy context from the previous newline character to the present character */
	  /* NOTE: Each line is only counted once - irrespective of number of occurances */
	  /* TODO: allocated only 100 bytes of space - if it exceeds, do a check and malloc as necessary */
	  memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	  (result_loc + res_idx)->line = line - 1;
	  res_idx += 1;
	  matched = 0;
	}

	/* Complicated way of avoiding control divergence to keep track of the previous new line occurance */
	out_before = out_before * (c != '\n') + i * (c == '\n');

	/* Need to remember whether some valid match occured on this line before - so || to not lose previous data */
	matched += re_matchp(pattern, (start + i));
  }

  /* There might be some matched string still waiting to find its ending newline character */
  if(matched){
	for(; (c = *(start + i) != '\n' && c != '\0'); i++);
	memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	(result_loc + res_idx)->line = line - 1;
  }
}

extern "C" void regex_match(char** file_names, file_info* info, int n_files, char* pattern){

  struct timeval start, end;
  gettimeofday(&start, NULL);

  /* re_print(re_pattern); */

  /* Copying file related data to device memory */
  char** device_contents;
  char** temp = (char**) malloc(n_files * sizeof(char*));
  hipMalloc(&device_contents, n_files * sizeof(char*));

  hipStream_t streams[n_files];
  for(int i = 0; i < n_files; i++){
	/* hipMalloc(&temp[i], MAX_FILE_SIZE * sizeof(char)); */
	/* hipMemcpy(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice); */
	/* hipMemcpy(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice); */

	hipStreamCreate(&streams[i]);
	hipHostRegister(info[i].mmap, info[i].size, 0);
	hipHostMalloc(&temp[i], info[i].size);
	hipMemcpyAsync(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice, streams[i]);
	hipMemcpyAsync(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice, streams[i]);
  }

  re_t re_pattern = re_compile(pattern);
  gettimeofday(&end, NULL);

  /* Creating an array of array of array of results: */
  res*** results;
  int* threads_size = (int*) malloc(n_files * sizeof(int));
  /* First pointer to index the file being grepped */
  hipMallocManaged(&results, n_files * sizeof(res**));
  for(int i = 0; i < n_files; i++){
	/* Second malloc to index the thread doing the computation */
	int n_chunks = info[i].size/CHUNK + 1;
	hipMallocManaged(&(results[i]), n_chunks * sizeof(res*));
	threads_size[i] = n_chunks;
	for(int j = 0; j < n_chunks; j++){
	  /* Third to index the result that the thread found */
	  /* TODO: Fourth to index the dynamic array for that result which will have a next pointer */
	  hipMallocManaged(&(results[i][j]), N_RESULTS * sizeof(res));
	  for(int k = 0; k < N_RESULTS; k++){
		hipMallocManaged(&(results[i][j][k].context), MAX_CONTEXT_SIZE);
	  }
	}
  }

  for(int i = 0; i < n_files; i++){
	if(threads_size[i] > MAX_THREADS_PER_BLOCK){
	  int n_blocks = threads_size[i]/MAX_THREADS_PER_BLOCK + 1;
	  regex_kernel <<< n_blocks, MAX_THREADS_PER_BLOCK, 0, streams[i] >>> (device_contents, results, re_pattern, i);
	}
	else{
	  regex_kernel <<< 1, threads_size[i], 0, streams[i] >>> (device_contents, results, re_pattern, i);
	}
	/* Unpinning the memory */
	hipHostUnregister(info[i].mmap);
  }

  hipDeviceSynchronize();


  /* printf("%s\n", hipGetErrorString(hipPeekAtLastError())); */
  /* printf("%d\n", results[0][0][0].line); */

  res result;
  for(int i = 0; i < n_files; i++){
	for(int j = 0; j < threads_size[i]; j++){
	  for(int k = 0; k < N_RESULTS; k++){
		result = results[i][j][k];
		if(result.line != 0)
		  printf("%s\n", result.context);
	  }
	}
  }

  hipFree(results);
  hipFree(device_contents);
  hipFree(re_pattern);

  printf("Kernel: %f\n", (end.tv_sec - start.tv_sec) * 1000 + (end.tv_usec - start.tv_usec)/(double)1000);
}

