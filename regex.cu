#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "regex.h"
#include "nfa_device.h"
}

#define MAX_FILE_SIZE 1 << 30
const int CHUNK = 5000;
const int MAX_CONTEXT_SIZE = 500;

typedef struct search_result {
  char* context;
  int line;
} res;

__global__ void regex_kernel(char** contents, res*** results, State* pattern, int file_no){
  int res_idx = 0;
  uint8_t valid;
  int line = 1;

  /* Local variables that keep track of the start and end of the context */
  /* TODO: out_before: needs to be initialized by going back until previous newline is found */
  int out_before = -1;
  uint8_t matched = 0;

  /* Read the ith file, check for pattern and write to result */
  char* start = &(contents[file_no][threadIdx.x * CHUNK]);
  res* result_loc = &(results[file_no][threadIdx.x][0]);
  char c;

  int i;
  for(i = 0; i < threadIdx.x * CHUNK && *(start-i) != '\n'; i++);
  out_before = -1 * i;

  for(i = 0; i < CHUNK && ((c = *(start + i)) != '\0'); i++){
	valid = 1;
	line += (c == '\n');
	if(matched && (c == '\n')){
	  /* Copy context from the previous newline character to the present character */
	  /* NOTE: Each line is only counted once - irrespective of number of occurances */
	  /* TODO: allocated only 100 bytes of space - if it exceeds, do a check and malloc as necessary */
	  memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	  (result_loc + res_idx)->line = line - 1;
	  res_idx += 1;
	  matched = 0;
	}
	/* Complicated way of avoiding control divergence to keep track of the previous new line occurance */
	out_before = out_before * (c != '\n') + i * (c == '\n');

	

	/* Need to remember whether some valid match occured on this line before - so || to not lose previous data */
	matched = matched || (valid != 0);
  }

  /* There might be some matched string still waiting to find its ending newline character */
  if(matched){
	for(; (c = *(start + i) != '\n'); i++);
	  memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	  (result_loc + res_idx)->line = line - 1;
  }
}

extern "C" void regex_match(char** file_names, file_info* info, int n_files, State* pattern){
  /* Copying file related data to device memory */
  char** device_contents;
  char** temp = (char**) malloc(n_files * sizeof(char*));
  hipMalloc(&device_contents, n_files * sizeof(char*));

  hipStream_t streams[n_files];
  for(int i = 0; i < n_files; i++){
	hipHostRegister(info[i].mmap, info[i].size, 0);
	hipMalloc(&temp[i], MAX_FILE_SIZE * sizeof(char));
	/* hipMemcpy(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice); */
	/* hipMemcpy(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice); */
	hipStreamCreate(&streams[i]);
	/* hipHostMalloc(&(info[i].mmap), (size_t) info[i].size); */
	/* hipHostMalloc(&temp[i], info[i].size); */
	hipMemcpyAsync(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice, streams[i]);
	hipMemcpyAsync(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice, streams[i]);
  }

  /* Copying the pattern to device memory */
  State* device_pattern;
  hipMalloc(&device_pattern, sizeof(State));
  hipMemcpy(device_pattern, pattern, sizeof(State), hipMemcpyHostToDevice);

  /* Creating an array of array of array of results: */
  res*** results;
  int* threads_size = (int*) malloc(n_files * sizeof(int));
  /* First pointer to index the file being grepped */
  hipMallocManaged(&results, n_files * sizeof(res**));
  for(int i = 0; i < n_files; i++){
	/* Second malloc to index the thread doing the computation */
	int n_chunks = info[i].size/CHUNK + 1;
	hipMallocManaged(&(results[i]), n_chunks * sizeof(res*));
	threads_size[i] = n_chunks;
	for(int j = 0; j < n_chunks; j++){
	  /* Third to index the result that the thread found */
	  /* TODO: Fourth to index the dynamic array for that result which will have a next pointer */
	  hipMallocManaged(&(results[i][j]), 50 * sizeof(res));
	  for(int k = 0; k < 50; k++){
		hipMallocManaged(&(results[i][j][k].context), MAX_CONTEXT_SIZE);
	  }
	}
  }

  for(int i = 0; i < n_files; i++){
	regex_kernel <<< 1, threads_size[i], 0, streams[i] >>> (device_contents, results, pattern, i);
	/* Unpinning the memory */
	hipHostUnregister(info[i].mmap);
  }

  hipDeviceSynchronize();

  res result;
  for(int i = 0; i < n_files; i++){
	for(int j = 0; j < threads_size[i]; j++){
	  for(int k = 0; k < 20; k++){
		result = results[i][j][k];
		if(result.line != 0)
		  printf("%s\n", result.context);
	  }
	}
  }
}
