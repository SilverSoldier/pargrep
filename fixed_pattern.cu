#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "fixed_pattern.h"
}

#define MAX_FILE_SIZE 1 << 30
const int CHUNK = 4 << 15;
const int MAX_CONTEXT_SIZE = 500;
const int N_RESULTS = 150;

typedef struct search_result {
  char* context;
  int line;
} res;

__global__ void fixed_pattern_kernel(char** contents, res*** results, const char* pattern, int file_no){
  int res_idx = 0;
  uint8_t valid;
  int line = 1;

  /* Local variables that keep track of the start and end of the context */
  /* TODO: out_before: needs to be initialized by going back until previous newline is found */
  int out_before = -1;
  uint8_t matched = 0;

  /* Read the ith file, check for pattern and write to result */
  char* start = &(contents[file_no][threadIdx.x * CHUNK]);
  res* result_loc = &(results[file_no][threadIdx.x][0]);
  char c;

  int i;
  for(i = 0; i < threadIdx.x * CHUNK && *(start-i) != '\n'; i++);
  out_before = -1 * i;

  for(i = 0; i < CHUNK && ((c = *(start + i)) != '\0'); i++){
	valid = 1;
	line += (c == '\n');
	if(matched && (c == '\n')){
	  /* Copy context from the previous newline character to the present character */
	  /* NOTE: Each line is only counted once - irrespective of number of occurances */
	  /* TODO: allocated only 100 bytes of space - if it exceeds, do a check and malloc as necessary */
	  memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	  (result_loc + res_idx)->line = line - 1;
	  res_idx += 1;
	  matched = 0;
	}
	/* Complicated way of avoiding control divergence to keep track of the previous new line occurance */
	out_before = out_before * (c != '\n') + i * (c == '\n');

	for(int j = 0; pattern[j] != '\0'; j++){
	  char c = *(start + i + j);
	  char p = pattern[j];
	  if(c == '\0')
		break;
	  /* Could break at this stage - not sure if there will be any gain due to control divergence */
	  valid &= (c == p);
	}
	/* Need to remember whether some valid match occured on this line before - so || to not lose previous data */
	matched = matched || (valid != 0);
  }

  /* There might be some matched string still waiting to find its ending newline character */
  if(matched){
	for(; (c = *(start + i) != '\n'); i++);
	memcpy((result_loc + res_idx)->context, (void*)(start + out_before+1), i - out_before - 1);
	(result_loc + res_idx)->line = line - 1;
  }
}

extern "C" void fixed_pattern_match(char** file_names, file_info* info, int n_files, char* pattern){
  /* Copying file related data to device memory */
  char** device_contents;
  char** temp = (char**) malloc(n_files * sizeof(char*));
  hipMalloc(&device_contents, n_files * sizeof(char*));

  hipStream_t streams[n_files];
  for(int i = 0; i < n_files; i++){
	hipMalloc(&temp[i], info[i].size * sizeof(char));
	hipMemcpy(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice);
	hipMemcpy(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice);

	/* hipHostRegister(info[i].mmap, info[i].size, 0); */
	/* hipStreamCreate(&streams[i]); */
	/* hipMemcpyAsync(temp[i], info[i].mmap, info[i].size, hipMemcpyHostToDevice, streams[i]); */
	/* hipMemcpyAsync(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice, streams[i]); */
	/* Unpinning the memory */
	/* hipHostUnregister(info[i].mmap); */
  }

  /* Copying the pattern to device memory */
  char* device_pattern;
  hipMalloc(&device_pattern, strlen(pattern));
  hipMemcpy(device_pattern, pattern, strlen(pattern), hipMemcpyHostToDevice);

  /* Creating an array of array of array of results: */
  res*** results;
  int* threads_size = (int*) malloc(n_files * sizeof(int));
  /* First pointer to index the file being grepped */
  hipMallocManaged(&results, n_files * sizeof(res**));
  for(int i = 0; i < n_files; i++){
	/* Second malloc to index the thread doing the computation */
	int n_chunks = info[i].size/CHUNK + 1;
	hipMallocManaged(&(results[i]), n_chunks * sizeof(res*));
	threads_size[i] = n_chunks;
	for(int j = 0; j < n_chunks; j++){
	  /* Third to index the result that the thread found */
	  /* TODO: third to index the dynamic array for that result which will have a next pointer */
	  hipMallocManaged(&(results[i][j]), N_RESULTS * sizeof(res));
	  for(int k = 0; k < N_RESULTS; k++){
		hipMallocManaged(&(results[i][j][k].context), MAX_CONTEXT_SIZE);
	  }
	}
  }

  for(int i = 0; i < n_files; i++){
	fixed_pattern_kernel <<< 1, threads_size[i] >>> (device_contents, results, device_pattern, i);
  }

  hipDeviceSynchronize();

  res result;
  for(int i = 0; i < n_files; i++){
	for(int j = 0; j < threads_size[i]; j++){
	  for(int k = 0; k < N_RESULTS; k++){
		result = results[i][j][k];
		if(result.line != 0)
		  printf("%s\n", result.context);
	  }
	}
  }

  hipFree(results);
  hipFree(device_contents);
  hipFree(device_pattern);
}

__global__ void test_kernel(int* A){
  A[threadIdx.x + blockIdx.x * blockDim.x] = 10;
}

extern "C" void test(){
  int* host_A = (int*) malloc(1000 * sizeof(int));
  int* device_A;
  hipMalloc((void**)&device_A, 1000 * sizeof(int));

  hipMemcpy(device_A, host_A, 1000 * sizeof(int), hipMemcpyHostToDevice);

  test_kernel <<< 1, 1000 >>> (device_A);

  hipMemcpy(host_A, device_A, 1000 * sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < 100; i++)
	if(host_A[i] != 10){
	  printf("Check failed.\n");
	  free(host_A);
	  hipFree(device_A);
	  return;
	}

  printf("Check passed.\n");
  free(host_A);
  hipFree(device_A);
}
