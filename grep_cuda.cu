#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "grep_cuda.h"
}

#define MAX_FILE_SIZE 1 << 30

__global__ void test(int* A){
  A[threadIdx.x + blockIdx.x * blockDim.x] = 10;
}

extern "C" void parallel_grep(){
  int* host_A = (int*) malloc(1000 * sizeof(int));
  int* device_A;
  hipMalloc((void**)&device_A, 1000 * sizeof(int));

  hipMemcpy(device_A, host_A, 1000 * sizeof(int), hipMemcpyHostToDevice);

  test <<< 1000, 1 >>> (device_A);

  hipMemcpy(host_A, device_A, 1000 * sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < 100; i++)
	if(host_A[i] != 10){
	  printf("Check failed.\n");
	  free(host_A);
	  hipFree(device_A);
	  return;
	}

  printf("Check passed.\n");
  free(host_A);
  hipFree(device_A);
}
