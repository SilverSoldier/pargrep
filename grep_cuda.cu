#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "grep_cuda.h"
}

#define MAX_FILE_SIZE 1 << 30

typedef struct search_result {
  char context[30];
  int line;
  int file;
} res;

__global__ void grep_kernel(char** contents, res** results, int* results_size, const char* pattern){
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  int res_idx = 0;
  int valid;
  int line = 1;
  /* Read the ith file, check for pattern and write to result */
  for(int i = 0; contents[idx][i] != '\0'; i++){
	valid = 1;
	line += (contents[idx][i] == '\n');
	for(int j = 0; pattern[j] != '\0'; j++){
	  char c = contents[idx][i+j];
	  char p = pattern[j];
	  if(c == '\0')
		break;
	  valid &= (c == p);
	}
	if(valid){
	  memcpy((void*)&(results[res_idx]->context[res_idx]), (void*)&(contents[idx][i-15]), 30);
	  results[res_idx]->context[29] = '\0';
	  results[idx][res_idx].file = idx;
	  results[idx][res_idx].line = line;
	  results_size[idx]++;
	  /*   res_idx++; */
	}
  }
}

extern "C" void parallel_grep(char** file_names, char** content, int n_files, char* pattern){
  /* Copying file related data to device memory */
  char** device_contents;
  char** temp = (char**) malloc(n_files * sizeof(char*));
  hipMalloc(&device_contents, n_files * sizeof(char*));
  for(int i = 0; i < n_files; i++){
	hipMalloc(&temp[i], MAX_FILE_SIZE * sizeof(char));
	/* Second read of data - This is wasteful */
	hipMemcpy(temp[i], content[i], strlen(content[i]) * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(device_contents + i, &(temp[i]), sizeof(char*), hipMemcpyHostToDevice);
  }

  /* Copying the pattern to device memory */
  char* device_pattern;
  hipMalloc(&device_pattern, strlen(pattern));
  hipMemcpy(device_pattern, pattern, strlen(pattern), hipMemcpyHostToDevice);

  /* Creating an array of array of results - one array for each thread doing computation */
  res** results;
  int* results_size;
  hipMallocManaged(&results, n_files * sizeof(res*));
  hipMallocManaged(&results_size, n_files * sizeof(int));
  for(int i = 0; i < n_files; i++){
	hipMallocManaged(&(results[i]), 1000 * sizeof(res));
	results_size[i] = 0;
  }

  grep_kernel <<< 1, n_files >>> (device_contents, results, results_size, device_pattern);

  hipDeviceSynchronize();

  for(int i = 0; i < n_files; i++)
	for(int j = 0; j < results_size[i]; j++){
	  res result = results[i][j];
	printf("%s:%d:%s", file_names[result.file], result.line, result.context);
	}
  printf("\n");
}

__global__ void test_kernel(int* A){
  A[threadIdx.x + blockIdx.x * blockDim.x] = 10;
}

extern "C" void test(){
  int* host_A = (int*) malloc(1000 * sizeof(int));
  int* device_A;
  hipMalloc((void**)&device_A, 1000 * sizeof(int));

  hipMemcpy(device_A, host_A, 1000 * sizeof(int), hipMemcpyHostToDevice);

  test_kernel <<< 1, 1000 >>> (device_A);

  hipMemcpy(host_A, device_A, 1000 * sizeof(int), hipMemcpyDeviceToHost);

  for(int i = 0; i < 100; i++)
	if(host_A[i] != 10){
	  printf("Check failed.\n");
	  free(host_A);
	  hipFree(device_A);
	  return;
	}

  printf("Check passed.\n");
  free(host_A);
  hipFree(device_A);
}
